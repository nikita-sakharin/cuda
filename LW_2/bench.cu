#include "hip/hip_runtime.h"
#include <errno.h>
#include <float.h>
#include <math.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

typedef signed char schar;
typedef unsigned char uchar;
typedef short shrt;
typedef unsigned short ushrt;
typedef unsigned uint;
typedef unsigned long ulong;
typedef long long llong;
typedef unsigned long long ullong;

typedef float flt;
typedef double dbl;
typedef long double ldbl;

#define exit_if(cnd_value, msg) \
    do { \
        if (cnd_value) \
        { \
            if (errno) \
                perror(msg); \
            else \
                fprintf(stderr, "error: %s\n", msg); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define max(a, b) ((a) >= (b) ? (a) : (b))
#define min(a, b) ((a) <= (b) ? (a) : (b))

#define cudaErrorCheck(error) \
    do { \
        hipError_t res = error; \
        if (res != hipSuccess) \
        { \
            fprintf(stderr, "cuda %s:%d error: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(res)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


#define DBL_CBRT_EPSILON (6.055454452393339060789E-6)

texture<uchar4, hipTextureType2D, hipReadModeElementType> tex;

__global__ void kernel(const uint tex_w, const uint tex_h,
    uchar4 * const __restrict__ dest, const uint dest_w, const uint dest_h)
{
    const int
        idxX = threadIdx.x + blockDim.x * blockIdx.x,
        idxY = threadIdx.y + blockDim.y * blockIdx.y,
        offsetX = blockDim.x * gridDim.x,
        offsetY = blockDim.y * gridDim.y;
    for (int i = idxX; i < dest_w; i += offsetX)
    {
        for (int j = idxY; j < dest_h; j += offsetY)
        {
            const dbl
                x = (i + 0.5) * tex_w / dest_w,
                y = (j + 0.5) * tex_h / dest_h;
            dbl
                x1 = floor(x - 0.5),
                x2 = ceil(x - 0.5),
                y1 = floor(y - 0.5),
                y2 = ceil(y - 0.5);
            const uchar4
                f11 = tex2D(tex, x1, y1),
                f12 = tex2D(tex, x1, y2),
                f21 = tex2D(tex, x2, y1),
                f22 = tex2D(tex, x2, y2);
            x1 += 0.5 - DBL_CBRT_EPSILON;
            x2 += 0.5 + DBL_CBRT_EPSILON;
            y1 += 0.5 - DBL_CBRT_EPSILON;
            y2 += 0.5 + DBL_CBRT_EPSILON;
            const dbl divides = (x2 - x1) * (y2 - y1);
            double4 f;
            f.x =
                f11.x * (x2 - x) * (y2 - y) +
                f12.x * (x2 - x) * (y - y1) +
                f21.x * (x - x1) * (y2 - y) +
                f22.x * (x - x1) * (y - y1);
            f.x /= divides;
            f.y =
                f11.y * (x2 - x) * (y2 - y) +
                f12.y * (x2 - x) * (y - y1) +
                f21.y * (x - x1) * (y2 - y) +
                f22.y * (x - x1) * (y - y1);
            f.y /= divides;
            f.z =
                f11.z * (x2 - x) * (y2 - y) +
                f12.z * (x2 - x) * (y - y1) +
                f21.z * (x - x1) * (y2 - y) +
                f22.z * (x - x1) * (y - y1);
            f.z /= divides;
            f.w =
                f11.w * (x2 - x) * (y2 - y) +
                f12.w * (x2 - x) * (y - y1) +
                f21.w * (x - x1) * (y2 - y) +
                f22.w * (x - x1) * (y - y1);
            f.w /= divides;
            dest[j * dest_w + i] = make_uchar4(f.x, f.y, f.z, f.w);
            fma(1.0, 2.0, 3.0);
        }
    }
}

int main(void)
{
    uint new_w, new_h, w, h;
    scanf("%u%u%u%u", &new_w, &new_h, &w, &h);

    uchar4 * const img = (uchar4 *) malloc(sizeof(uchar4) * max(w * h, new_w * new_h));
    exit_if(!img, "malloc()");

    hipChannelFormatDesc channel = hipCreateChannelDesc<uchar4>();
    cudaErrorCheck(hipGetLastError());

    hipArray *device_array;
    cudaErrorCheck(hipMallocArray(&device_array, &channel, w, h));
    cudaErrorCheck(hipMemcpyToArray(device_array, 0, 0, img, sizeof(uchar4) * w * h,
        hipMemcpyHostToDevice));

    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = channel;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;
    cudaErrorCheck(hipBindTextureToArray(tex, device_array, channel));

    uchar4 *dev_img;
    cudaErrorCheck(hipMalloc(&dev_img, sizeof(uchar4) * new_w * new_h));

    hipEvent_t start, stop;
    cudaErrorCheck(hipEventCreate(&start));
    cudaErrorCheck(hipEventCreate(&stop));
    cudaErrorCheck(hipEventRecord(start, 0));

    kernel<<<dim3(32U, 32U), dim3(32U, 32U)>>>(w, h, dev_img, new_w, new_h);
    cudaErrorCheck(hipGetLastError());

    cudaErrorCheck(hipEventRecord(stop, 0));
    cudaErrorCheck(hipEventSynchronize(stop));

    flt time;
    cudaErrorCheck(hipEventElapsedTime(&time, start, stop));
    cudaErrorCheck(hipEventDestroy(start));
    cudaErrorCheck(hipEventDestroy(stop));
    printf("time = %f\n", time);

    cudaErrorCheck(hipMemcpy(img, dev_img, sizeof(uchar4) * new_w * new_h,
        hipMemcpyDeviceToHost));

    cudaErrorCheck(hipUnbindTexture(tex));
    cudaErrorCheck(hipFreeArray(device_array));
    cudaErrorCheck(hipFree(dev_img));
    free(img);

    return 0;
}
